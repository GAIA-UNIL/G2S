#include "hip/hip_runtime.h"
/*
 * G2S
 * Copyright (C) 2018, Mathieu Gravey (gravey.mathieu@gmail.com) and UNIL (University of Lausanne)
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

//To use only for debugging purpose

#include <cstring>
#include "NvidiaGPUAcceleratorDevice.hpp"
#include "sharedMemoryManager.hpp"
#include "utils.hpp"
#include <algorithm>

#define PARTIAL_FFT

#ifndef FFTW_PLAN_OPTION
	//FFTW_PATIENT
	#define FFTW_PLAN_OPTION FFTW_ESTIMATE
#endif
// #if __cilk
// 	#define fillVectorized(name, begin, amount, value) name[begin:amount]=value;
// #else
	#define fillVectorized(name, begin, amount, value) std::fill(name+begin,name+begin+amount,value);
// #endif


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
 #define gpuErrchk(ans) { ans; }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

inline void gpuAssert(hipblasStatus_t code, const char *file, int line, bool abort=true)
{
	if (code != HIPBLAS_STATUS_SUCCESS) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", _cudaGetErrorEnum(code), file, line);
		if (abort) exit(code);
	}
}

static const char *_cudaGetErrorEnum(hipfftResult error)
{
    switch (error)
    {
        case HIPFFT_SUCCESS:
            return "HIPFFT_SUCCESS";

        case HIPFFT_INVALID_PLAN:
            return "HIPFFT_INVALID_PLAN";

        case HIPFFT_ALLOC_FAILED:
            return "HIPFFT_ALLOC_FAILED";

        case HIPFFT_INVALID_TYPE:
            return "HIPFFT_INVALID_TYPE";

        case HIPFFT_INVALID_VALUE:
            return "HIPFFT_INVALID_VALUE";

        case HIPFFT_INTERNAL_ERROR:
            return "HIPFFT_INTERNAL_ERROR";

        case HIPFFT_EXEC_FAILED:
            return "HIPFFT_EXEC_FAILED";

        case HIPFFT_SETUP_FAILED:
            return "HIPFFT_SETUP_FAILED";

        case HIPFFT_INVALID_SIZE:
            return "HIPFFT_INVALID_SIZE";

        case HIPFFT_UNALIGNED_DATA:
            return "HIPFFT_UNALIGNED_DATA";
    }

    return "<unknown>";
}

inline void gpuAssert(hipfftResult code, const char *file, int line, bool abort=true)
{
	if (code != HIPFFT_SUCCESS) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", _cudaGetErrorEnum(code), file, line);
		if (abort) exit(code);
	}
}


/* ------------- start kernels -----------------*/

__host__ __device__ static __inline__  hipComplex cuCfmafAlpha( hipComplex x, hipComplex y, hipComplex d, float alpha)
{
	float real_res;
	float imag_res;
	
	real_res = (hipCrealf(x) *  hipCrealf(y))*alpha + hipCrealf(d);
	imag_res = (hipCrealf(x) *  hipCimagf(y))*alpha + hipCimagf(d);
			
	real_res = -(hipCimagf(x) * hipCimagf(y))*alpha  + real_res;  
	imag_res =  (hipCimagf(x) *  hipCrealf(y))*alpha + imag_res;          
	 
	return make_hipComplex(real_res, imag_res);
}

__global__ void complexAddAlphaxCxD(hipFloatComplex* dst, const hipFloatComplex* C, const hipFloatComplex* D, const float alpha, const unsigned int size){

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < size) dst[i]=cuCfmafAlpha(C[i],D[i],dst[i],alpha);
}

__global__ void removeBorder(float* dst, const unsigned int size, const unsigned int delta, const unsigned int blockSize, const float remplace){

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int blockId = blockIdx.y * blockDim.y + threadIdx.y;
	int pos = i * delta + blockId;
	// if (pos < size && blockId < blockSize){
	// 	dst[pos]=remplace;
	// }
}

__global__ void fma(float* realSpace, const unsigned int size,  const float alpha,  const float delta){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < size)
		realSpace[i]=fmaf(realSpace[i],alpha,delta);
}

__global__ void compensateMissingDatakernel(float* errosArray, float* crossErrosArray, const unsigned int size, float val){
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	if (j < size)
	{
		errosArray[j]=-std::fabs(errosArray[j]/(crossErrosArray[j]*crossErrosArray[j]*crossErrosArray[j]*crossErrosArray[j]));
		if(crossErrosArray[j]==0.0f) errosArray[j]=val;
	}
}

__global__ void copyAndRemove(float* errosArray, unsigned int*  _encodedPosition_d,float* _mismatch_d,const unsigned int i, const float val){

	_encodedPosition_d[i]-=1;
	if(_mismatch_d){
		_mismatch_d[i]=errosArray[_encodedPosition_d[i]];
	}
	errosArray[_encodedPosition_d[i]]=val;
}	

/* ------------- end kernels -----------------*/


#define cufftPlan(p, rank, n, type) hipfftPlanMany(p, rank, n, 0, 1, 1,  0, 1, 1, type, 1)

NvidiaGPUAcceleratorDevice::NvidiaGPUAcceleratorDevice(int deviceId, SharedMemoryManager* sharedMemoryManager,std::vector<g2s::OperationMatrix> coeficientMatrix,
	unsigned int threadRatio, bool withCrossMesurement, bool circularTI)
	:AcceleratorDevice( sharedMemoryManager, coeficientMatrix, threadRatio, withCrossMesurement, circularTI)
{
	_deviceType=DT_gpuCuda;
	// int chip,core;
	// g2s::rdtscp(&chip, &core);
	int numberOfDevice;
	gpuErrchk(hipGetDeviceCount(&numberOfDevice));
	_deviceID=deviceId%numberOfDevice;
	hipSetDevice(_deviceID);
	
	hipStreamCreate(&_cudaLocalStream);
	//_cudaLocalStream=0;
	gpuErrchk(hipEventCreate(&_cudaEventFinal));
	#pragma omp critical (createDevices)
	{
		_sharedMemoryManager->addDevice(this);
	}
	initDim();
	gpuErrchk(hipblasCreate(&_cublasHandle));
	gpuErrchk(hipblasSetStream(_cublasHandle, _cudaLocalStream));
	gpuErrchk(hipblasSetPointerMode(_cublasHandle, HIPBLAS_POINTER_MODE_DEVICE));

	gpuErrchk( hipMalloc(&_frenquencySpaceInput,_fftSpaceSize * sizeof(hipfftComplex)));

	for (size_t i = 0; i < _coeficientMatrix.size(); ++i)
	{
		hipfftComplex* ptrCplx;
		gpuErrchk( hipMalloc(&ptrCplx,_fftSpaceSize * sizeof(hipfftComplex)));
		_frenquencySpaceOutputArray.push_back(ptrCplx);
		float* ptrReal;
		gpuErrchk( hipMalloc(&ptrReal,_realSpaceSize* sizeof(dataType)));
		_realSpaceArray.push_back(ptrReal);
	}

	std::vector<int> reverseFftSize(_fftSize.begin(),_fftSize.end());
	std::reverse(reverseFftSize.begin(),reverseFftSize.end());
	{
		gpuErrchk(cufftPlan(&_pInv,reverseFftSize.size(), reverseFftSize.data(),HIPFFT_C2R));
		gpuErrchk(hipfftSetStream(_pInv,_cudaLocalStream));

		gpuErrchk(cufftPlan(&_p,reverseFftSize.size(), reverseFftSize.data(),HIPFFT_R2C));
		gpuErrchk(hipfftSetStream(_p,_cudaLocalStream));

		if(_fftSize.size()>1){
			unsigned reducedSize=1;

			for (size_t i = 1; i < _fftSize.size()-1; ++i)
			{
				reducedSize*=_fftSize[i];
			}

			_reducedFftSize=reducedSize*(_fftSize.front()/2+1);
			_reducedRealSize=reducedSize*(_fftSize.front());

			gpuErrchk(cufftPlan(&_pPatchL,reverseFftSize.size()-1, reverseFftSize.data()+1,HIPFFT_R2C));
			gpuErrchk(hipfftSetStream(_pPatchL,_cudaLocalStream));

			gpuErrchk(hipfftPlanMany(&_pPatchM, 1, reverseFftSize.data(), reverseFftSize.data(),
				_reducedFftSize, 1, reverseFftSize.data(),
				_reducedFftSize, 1, HIPFFT_R2C, _reducedFftSize));
			gpuErrchk(hipfftSetStream(_pPatchM,_cudaLocalStream));

		}

	}
	hipEventRecord(_cudaEventFinal,_cudaLocalStream);
	hipEventSynchronize(_cudaEventFinal);
}

NvidiaGPUAcceleratorDevice::~NvidiaGPUAcceleratorDevice(){
	_sharedMemoryManager->removeDevice(this);
	gpuErrchk(hipfftDestroy(_pInv));
	gpuErrchk(hipfftDestroy(_p));

	if(_fftSize.size()>1){
		gpuErrchk(hipfftDestroy(_pPatchM));
		gpuErrchk(hipfftDestroy(_pPatchL));
	}

	gpuErrchk(hipFree(_frenquencySpaceInput));
	for (size_t i = 0; i < _frenquencySpaceOutputArray.size(); ++i)
	{
		gpuErrchk(hipFree(_frenquencySpaceOutputArray[i]));
		_frenquencySpaceOutputArray[i]=nullptr;
	}
	for (size_t i = 0; i < _realSpaceArray.size(); ++i)
	{
		gpuErrchk(hipFree(_realSpaceArray[i]));
		_realSpaceArray[i]=nullptr;
	}

	if(_encodedPosition_d){
		gpuErrchk(hipFree(_encodedPosition_d));
		gpuErrchk(hipFree(_mismatch_d));
		_encodedPosition_d=nullptr;
		_mismatch_d=nullptr;
		_encodedPosition_dSize=0;	
	}

	gpuErrchk(hipEventDestroy(_cudaEventFinal));

	gpuErrchk(hipblasDestroy(_cublasHandle));
	if(_cudaLocalStream!=0)
		gpuErrchk(hipStreamDestroy(_cudaLocalStream));
}

std::vector<g2s::spaceFrequenceMemoryAddress> NvidiaGPUAcceleratorDevice::allocAndInitSharedMemory(std::vector<void* > srcMemoryAdress, std::vector<unsigned> srcSize, std::vector<unsigned> fftSize){
	
	//fprintf(stderr, "alloc shared memory CPU\n");
	hipError_t hipError_t;
	unsigned fftSpaceSize=1;
	unsigned realSpaceSize=1;

	fftSpaceSize*=fftSize.front()/2+1;
	realSpaceSize*=fftSize.front();

	for (size_t i = 1; i < fftSize.size(); ++i)
	{
		fftSpaceSize*=fftSize[i];
		realSpaceSize*=fftSize[i];
	}

	std::vector<int> reverseFftSize(fftSize.begin(),fftSize.end());
	std::reverse(reverseFftSize.begin(),reverseFftSize.end());

	std::vector<g2s::spaceFrequenceMemoryAddress> sharedMemory;
	for (size_t i = 0; i < srcMemoryAdress.size(); ++i)
	{
		g2s::spaceFrequenceMemoryAddress sharedMemoryAdress;
		gpuErrchk(hipMalloc(&sharedMemoryAdress.space, realSpaceSize * sizeof(dataType)));
		gpuErrchk(hipMemcpyAsync(sharedMemoryAdress.space, srcMemoryAdress[i], realSpaceSize * sizeof(dataType), hipMemcpyHostToDevice, _cudaLocalStream));
		gpuErrchk(hipMalloc(&sharedMemoryAdress.fft, fftSpaceSize * sizeof(hipfftComplex)));
		
		sharedMemory.push_back(sharedMemoryAdress);

		hipfftHandle p;
			
		hipfftResult cufftError;
		gpuErrchk(cufftPlan(&p,reverseFftSize.size(), reverseFftSize.data(),HIPFFT_R2C));
		gpuErrchk(hipfftSetStream(p,_cudaLocalStream));
		//FFTW_PRECISION(plan_dft_r2c)(reverseFftSize.size(), reverseFftSize.data(), (dataType*)sharedMemoryAdress.space, (hipfftComplex*)sharedMemoryAdress.fft, FFTW_ESTIMATE);
		gpuErrchk(hipfftExecR2C(p, (dataType*)sharedMemoryAdress.space, (hipfftComplex*)sharedMemoryAdress.fft));
		gpuErrchk(hipfftDestroy(p));
	}
	return sharedMemory;

}

std::vector<g2s::spaceFrequenceMemoryAddress> NvidiaGPUAcceleratorDevice::freeSharedMemory(std::vector<g2s::spaceFrequenceMemoryAddress> sharedMemoryAdress){
	for (size_t i = 0; i < sharedMemoryAdress.size(); ++i)
	{
		gpuErrchk(hipFree(sharedMemoryAdress[i].space));
		gpuErrchk(hipFree(sharedMemoryAdress[i].fft));
	}
	sharedMemoryAdress.clear();
	return sharedMemoryAdress;
}

//compute function

unsigned NvidiaGPUAcceleratorDevice::getArraySize(){
	return _realSpaceSize;
}

float NvidiaGPUAcceleratorDevice::getValueAtPosition(unsigned arrayIndex, unsigned index){
	float tmp;
	gpuErrchk(hipMemcpyAsync(_realSpaceArray[arrayIndex]+index, &tmp, sizeof(float), hipMemcpyDeviceToHost, _cudaLocalStream));
	gpuErrchk(hipStreamSynchronize(_cudaLocalStream));
	return tmp;
}

float NvidiaGPUAcceleratorDevice::getErrorAtPosition(unsigned index){	
	float tmp;
	gpuErrchk(hipMemcpyAsync(_realSpaceArray.front()+index, &tmp, sizeof(float), hipMemcpyDeviceToHost, _cudaLocalStream));
	gpuErrchk(hipStreamSynchronize(_cudaLocalStream));
	return tmp;
}

float NvidiaGPUAcceleratorDevice::getCroossErrorAtPosition(unsigned index){	
	float tmp;
	gpuErrchk(hipMemcpyAsync(_realSpaceArray.back()+index, &tmp, sizeof(float), hipMemcpyDeviceToHost, _cudaLocalStream));
	gpuErrchk(hipStreamSynchronize(_cudaLocalStream));
	return tmp;
}

unsigned NvidiaGPUAcceleratorDevice::getErrorsArraySize(){
	return _realSpaceSize;
}

unsigned NvidiaGPUAcceleratorDevice::cvtIndexToPosition(unsigned index){
	
	unsigned position=0;
	unsigned divFactor=_realSpaceSize;
	for (int i = int(_fftSize.size()-1); i>=0; --i)
	{
		divFactor/=_fftSize[i];
		position=position*_srcSize[i] + ((_fftSize[i]-(index/(divFactor))%_fftSize[i]-_min[i]-1)+_srcSize[i])%_srcSize[i];
	}

	return position;
}

unsigned NvidiaGPUAcceleratorDevice::cvtPositionToIndex(unsigned position){

	unsigned index=0;
	unsigned divFactor=1;
	for (int i = int(_fftSize.size()-1); i>=0; --i)
	{
		divFactor*=_srcSize[i];
	}
	for (int i = int(_fftSize.size()-1); i>=0; --i)
	{
		divFactor/=_srcSize[i];
		index=index*_fftSize[i] + (_fftSize[i]-(position/(divFactor)+_min[i])%_srcSize[i]-1);
	}
	return index;
}

void NvidiaGPUAcceleratorDevice::zerosFrenquencySpaceOutputArray(unsigned layer){
	gpuErrchk(hipMemsetAsync(_frenquencySpaceOutputArray[layer], 0, _fftSpaceSize * sizeof(hipfftComplex), _cudaLocalStream ));
}

void NvidiaGPUAcceleratorDevice::computeFreqMismatchMap(std::vector<std::vector<int> > neighborArray, std::vector<std::vector<float> >  &neighborValueArrayVector){
	
	for (unsigned int var = 0; var <_coeficientMatrix[0].getNumberOfVariable() ; ++var)
	{
		bool lines[_fftSize.back()];

		bool needTobeComputed=false;
		for (size_t dataArrayIndex = 0; dataArrayIndex < _coeficientMatrix.size(); ++dataArrayIndex)
		{
			needTobeComputed|=_coeficientMatrix[dataArrayIndex].needVariableAlongB(var);
		}
		if(!needTobeComputed) return;

		gpuErrchk(hipMemsetAsync(_realSpaceArray[0],0,sizeof(dataType) * _realSpaceSize, _cudaLocalStream ));
		gpuErrchk(hipMemsetAsync(_frenquencySpaceInput,0,_fftSpaceSize * sizeof(hipfftComplex), _cudaLocalStream ));

		for (size_t i = 0; i < neighborArray.size(); ++i)
		{
			gpuErrchk(hipMemcpyAsync(_realSpaceArray[0] + index(neighborArray[i]), &neighborValueArrayVector[i][var], sizeof(float), hipMemcpyHostToDevice, _cudaLocalStream));
			lines[neighborArray[i].back()]=true;
		}

		bool patialFFT=false;

		#ifdef PARTIAL_FFT
		//patialFFT=true;
		#endif

		hipfftResult cufftError;
		if(patialFFT && (_fftSize.size()>1)){
			
			for (unsigned int i = 0; i < _fftSize.back(); ++i)
			{
				if(lines[i]){
					gpuErrchk(hipfftExecR2C(_pPatchL, _realSpaceArray[0]+i*_reducedRealSize, _frenquencySpaceInput+i*_reducedFftSize));
				}
			}
			gpuErrchk(hipfftExecC2C(_p, _frenquencySpaceInput, _frenquencySpaceInput, HIPFFT_FORWARD));
		}else{
			gpuErrchk(hipfftExecR2C(_p, _realSpaceArray[0], _frenquencySpaceInput));
		}


		for (size_t dataArrayIndex = 0; dataArrayIndex < _coeficientMatrix.size(); ++dataArrayIndex)
		{
			for (unsigned int varA = 0; varA < _coeficientMatrix[dataArrayIndex].getNumberOfVariable(); ++varA)
			{
				float localCoef=_coeficientMatrix[dataArrayIndex].getVariableAt(varA,var);
				if (localCoef!=0.f)
				{
					complexAddAlphaxCxD<<<(_fftSpaceSize+255)/256, 256,0, _cudaLocalStream >>>(_frenquencySpaceOutputArray[dataArrayIndex],(const hipFloatComplex *)_srcCplx[varA].fft, (const hipFloatComplex *)_frenquencySpaceInput, localCoef, _fftSpaceSize);
					gpuErrchk(hipPeekAtLastError());
				}
			}
		}
	}
}


void NvidiaGPUAcceleratorDevice::computeRealMissmatchAndRemoveWrongPattern(float* delta0)
{
	const short chunk=32;
	for (size_t dataArrayIndex = 0; dataArrayIndex < _coeficientMatrix.size(); ++dataArrayIndex)
	{
		
		gpuErrchk(hipfftExecC2R(_pInv, _frenquencySpaceOutputArray[dataArrayIndex], _realSpaceArray[dataArrayIndex]));

		dataType* realSpace= _realSpaceArray[dataArrayIndex];

		//Remove fobidden/wrong value
		if (!_circularTI)
		{
			for (size_t i = 0; i < _fftSize.size(); ++i)
			{

				unsigned blockSize=1;
				for (size_t j = 0; j < i; ++j)
				{
					blockSize*=_fftSize[j];
				}
				blockSize*=_fftSize[i]-(_srcSize[i]-(_max[i]-_min[i]));

				unsigned delta=1;
				for (size_t j = 0; j <= i; ++j)
				{
					delta*=_fftSize[j];
				}

				dim3 grid((_realSpaceSize/delta+(chunk-1))/chunk,(blockSize+(chunk-1))/chunk,1);
				dim3 block(chunk,chunk,1);
				
				//fprintf(stderr, "blockSize ==> %d, %d ==> %d, %d \n", (_realSpaceSize/delta+(chunk-1))/chunk, (blockSize+(chunk-1))/chunk, delta, blockSize );

				if(blockSize!=0)
					removeBorder<<<grid, block, 0, _cudaLocalStream >>>(realSpace,_realSpaceSize,delta,blockSize,-INFINITY);
				gpuErrchk(hipPeekAtLastError());
			}
		}

		if(_trueMismatch && !_crossMesurement) // correct value needed
		{
			fma<<<(_realSpaceSize+255)/256, 256, 0, _cudaLocalStream >>>(realSpace, _realSpaceSize, 1./_realSpaceSize, delta0[dataArrayIndex]);
			gpuErrchk(hipPeekAtLastError());
		}
	}
}


void NvidiaGPUAcceleratorDevice::maskLayerWithVariable(unsigned layer, unsigned variable){

	int deltaCross=0;
	for (int k = int(_min.size())-1; k >=0; k--)
	{
		deltaCross=deltaCross*_fftSize[k]+_min[k];
	}
	int convertedVariable=0;
	int tmp=variable;
	for (unsigned int var = 0; var <_coeficientMatrix[layer].getNumberOfVariable() ; ++var)
	{
		tmp-=_coeficientMatrix[layer].needVariableAlongA(var);
		if(tmp<0)
		{
			convertedVariable=var;
			break;
		}
	}
	//TODO : to remove missing data point
	/*for (unsigned int i = 0; i < _realSpaceSize; ++i){
		_realSpaceArray[layer][i]*=((dataType*)_srcCplx[convertedVariable].space)[(i+deltaCross)%_realSpaceSize];

		//-((1.f-[j])*1.1f)*FLT_MAX);
	}*/
}

void NvidiaGPUAcceleratorDevice::setValueInErrorArray(unsigned position, float value){
	
	float* errosArray=_realSpaceArray.front();
	errosArray[position]=value;
}

void NvidiaGPUAcceleratorDevice::compensateMissingData(){
	
	float* errosArray=_realSpaceArray.front();
	float* crossErrosArray=_realSpaceArray.back();
	unsigned sizeArray=getErrorsArraySize();

	compensateMissingDatakernel<<<(_realSpaceSize+255)/256, 256, 0, _cudaLocalStream >>>(errosArray, crossErrosArray, _realSpaceSize,-INFINITY);
	gpuErrchk(hipPeekAtLastError());
}

void NvidiaGPUAcceleratorDevice::searchKBigest(float* errors,unsigned *encodedPosition, unsigned extendK, float seed){
	
	float* errosArray=_realSpaceArray.front();
	unsigned sizeArray=getErrorsArraySize();

	if(_encodedPosition_dSize<extendK)
	{
		if(_encodedPosition_d!=nullptr)
		{
			gpuErrchk(hipFree(_encodedPosition_d));
			gpuErrchk(hipFree(_mismatch_d));
			_encodedPosition_d=nullptr;
			_encodedPosition_dSize=0;
		}
		gpuErrchk(hipMalloc(&_encodedPosition_d,extendK*sizeof(unsigned int)));
		gpuErrchk(hipMalloc(&_mismatch_d,extendK*sizeof(unsigned int)));
		_encodedPosition_dSize=extendK;
	};

	for (int i = 0; i < extendK; ++i)
	{

		gpuErrchk(hipblasIsamin(_cublasHandle, sizeArray, errosArray, 1, ((int*)_encodedPosition_d)+i));
		copyAndRemove<<<1,1,0,_cudaLocalStream>>>(errosArray,_encodedPosition_d,_mismatch_d,i,-INFINITY);
		gpuErrchk(hipPeekAtLastError());
	}

	gpuErrchk(hipMemcpyAsync(encodedPosition, _encodedPosition_d, extendK*sizeof(unsigned int), hipMemcpyDeviceToHost, _cudaLocalStream));
	gpuErrchk(hipMemcpyAsync(errors, _mismatch_d, extendK*sizeof(float), hipMemcpyDeviceToHost, _cudaLocalStream));
	gpuErrchk(hipEventRecord(_cudaEventFinal,_cudaLocalStream));
	gpuErrchk(hipEventSynchronize(_cudaEventFinal));
	// gpuErrchk(hipDeviceSynchronize());
	// for (int i = 0; i < extendK; ++i)
	// {
	// 	encodedPosition[i]=1;
	// 	errors[i]=0;
	// }
	

}
